#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../inc/hist.h"

#define CHECK(call)                                                     \
    {                                                                   \
        const hipError_t error = call;                                 \
        if (error != hipSuccess)                                       \
            {                                                           \
                fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);  \
                fprintf(stderr, "code: %d, reason: %s\n", error,        \
                        hipGetErrorString(error));                     \
                exit(1);                                                \
            }                                                           \
    }
    
void runOnGpu(const unsigned char* colors, unsigned int* buckets, 
                unsigned int len, unsigned int rows, unsigned int cols, 
                void(*gpuFunc)(unsigned char*, unsigned int*, unsigned int, unsigned int, unsigned int)
    ) {
    unsigned char* d_colors;
    int* d_buckets;
    CHECK(hipMalloc(&d_colors, sizeof(unsigned char) * len));
    CHECK(hipMalloc(&d_buckets, sizeof(unsigned int) * 4* 256));
    CHECK(hipMemcpy(d_colors, colors, sizeof(unsigned char) * len, hipMemcpyHostToDevice));
    dim3 grid, block;
    block.x = 32;
    block.y = 32;
    grid.x = (rows - 1) / block.x + 1;
    grid.y = (cols - 1) / block.y + 1;
    /* printf("%d - %d\n", grid.x, grid.y); */
    (*gpuFunc)<<<grid, block>>>(d_colors, d_buckets, len, rows, cols);
    CHECK(hipMemcpy(buckets, d_buckets, sizeof(unsigned int) * 256, hipMemcpyDeviceToHost));
    CHECK(hipFree(d_colors));
    CHECK(hipFree(d_buckets));
}


__global__ void gpuNaive(unsigned char* colors, unsigned int* buckets, unsigned int len, unsigned int rows, unsigned int cols) {
    printf("Using naive GPU implementation\n");
    int i = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * blockIdx.x + blockDim.x * blockDim.y * blockIdx.y * gridDim.x;
    if (i < len) {
        // get wether rgb or alpha value 
        unsigned int color = i % 4;
        unsigned int entry = 256*color + colors[i];
	atomicAdd(&buckets[entry], 1);
    }
}

__global__ void gpuGood(unsigned char* colors, unsigned int* buckets, unsigned int len, unsigned int rows, unsigned int cols) {
    printf("Using good GPU implementation\n");
    // TODO
    int i = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * blockIdx.x + blockDim.x * blockDim.y * blockIdx.y * gridDim.x;
    if (i < len) {
        // get wether rgb or alpha value 
        unsigned int color = i % 4;
        unsigned int entry = 256*color + colors[i];
	atomicAdd(&buckets[entry], 1);
    }
}
