#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "../inc/hist.h"
#include <sys/time.h>

#define CHECK(call)                                                     \
    {                                                                   \
        const hipError_t error = call;                                 \
        if (error != hipSuccess)                                       \
            {                                                           \
                fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);  \
                fprintf(stderr, "code: %d, reason: %s\n", error,        \
                        hipGetErrorString(error));                     \
                exit(1);                                                \
            }                                                           \
    }
    
__global__ void gpuNaive(unsigned char* colors, unsigned int* buckets, unsigned int len, unsigned int rows, unsigned int cols) {
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int i = (iy * rows + ix)*4;
    if (i < len) {
        // get wether rgb or alpha value 
        unsigned int entry =  colors[i];
        atomicAdd(&buckets[entry], 1);
        entry = 256   + colors[i+1];
        atomicAdd(&buckets[entry], 1);
        entry = 256*2 + colors[i+2];
        atomicAdd(&buckets[entry], 1);
        entry = 256*3 + colors[i+3];
        atomicAdd(&buckets[entry], 1);
    }
}

__global__ void gpuGood_Block(unsigned char* colors, unsigned int* buckets, unsigned int len, unsigned int rows, unsigned int cols) {
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int i = (iy * blockDim.x * gridDim.x + ix);

    if (i < len) {
        //unsigned int offset = blockIdx.y * gridDim.x + blockIdx.x;
        //offset *= 4*256;
        // get wether rgb or alpha value 
        //unsigned int entry = offset+ colors[i];
        unsigned int entry = (i%4)*256 + colors[i];
	atomicAdd(&buckets[entry], 1);
        //entry = offset + 256   + colors[i+1];
        //atomicAdd(&buckets[entry], 1);
        //entry = offset + 256*2 + colors[i+2];
        //atomicAdd(&buckets[entry], 1);
        //entry = offset + 256*3 + colors[i+3];
        //atomicAdd(&buckets[entry], 1);
    }
}

__global__ void gpuGood_MergeBlocks(unsigned int* buckets, unsigned int blockcnt) {
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
    //unsigned int i = iy * 256 + ix;
    unsigned int i = (iy * blockDim.x * gridDim.x + ix); 
    
    /*
    if (i < 4*256) {

        //unsigned int offset = blockIdx.y * gridDim.x + blockIdx.x;
        //offset *= 4*256;
        for(unsigned int j=1; j < blockcnt; j++)
        {
            unsigned int entry = i+ j *4*256;
            //atomicAdd(&buckets[i], buckets[entry]);
	    buckets[i] += buckets[entry];
        }
    }*/
    
	    atomicAdd(&buckets[i%(256*4)], buckets[i+(256*4)]);
}

double runOnGpu(const unsigned char* colors, unsigned int* buckets, 
                unsigned int len, unsigned int rows, unsigned int cols, 
                unsigned char compute_function
    ) {
    if(compute_function > 2)
    {
        return -1 ;
    }

    struct timeval start, end;
    gettimeofday(&start,NULL);

    unsigned char* d_colors;
    unsigned int* d_buckets;
    CHECK(hipMalloc(&d_colors, sizeof(unsigned char) * len));
    CHECK(hipMemcpy(d_colors, colors, sizeof(unsigned char) * len, hipMemcpyHostToDevice));
        
    
    if(compute_function == 1)
    {
        dim3 grid, block;
        block.x = 256;
        block.y = 1;
        grid.x = ceil((double)(rows*cols)/ block.x); 
        grid.y = 1;
        
	CHECK(hipMalloc(&d_buckets, sizeof(unsigned int) * 256*4));
        printf("Using naive GPU implementation\n");
        gettimeofday(&start,NULL);
        gpuNaive<<<grid, block>>>(d_colors, d_buckets, len, rows, cols );
        gettimeofday(&end,NULL);
    
        CHECK(hipMemcpy(buckets, d_buckets, sizeof(unsigned int) *256*4, hipMemcpyDeviceToHost));
        CHECK(hipFree(d_buckets));
    }
    else
    {

        dim3 grid, block;
        block.x = 256;
        block.y = 4;
        grid.x = ceil((double)(rows*cols)/ block.x ); 
        grid.y = 1;
        
	CHECK(hipMalloc(&d_buckets, sizeof(unsigned int) * 256*4 * grid.x ));
        printf("Using good GPU implementation\n");
        gettimeofday(&start,NULL);
    
        gpuGood_Block<<<grid, block>>>(d_colors, d_buckets, len, rows, cols);
        unsigned int blockCnt = grid.x;
        //block.x = 256;
        //block.y = 4;
        grid.x -= 1; 
        //grid.y = 1;
       	//hipDeviceSynchronize(); 
        gpuGood_MergeBlocks<<<grid, block>>>(d_buckets, blockCnt);
        gettimeofday(&end,NULL);

        CHECK(hipMemcpy(buckets, d_buckets, sizeof(unsigned int) *256*4, hipMemcpyDeviceToHost));
        CHECK(hipFree(d_buckets));
    }
    CHECK(hipFree(d_colors));

    double start_seconds = ((double)start.tv_sec + (double)start.tv_usec*1.e-6);
    double end_seconds = ((double)end.tv_sec + (double)end.tv_usec*1.e-6);
    return end_seconds - start_seconds;

}


