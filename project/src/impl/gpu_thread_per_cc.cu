#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

extern "C" {
#include "impl.h"
}
#include "cuda_helpers.h"

#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void populateGraph(dense_graph *d_graph, dense_node *d_nodes,
                              dense_edge *d_edges, void *current_base) {
  int node = blockIdx.x * blockDim.x * blockDim.y + threadIdx.x * blockDim.y +
             threadIdx.y;
  if (node >= d_graph->num_nodes)
    return;
  if (node == 0) {
    d_graph->nodes = d_nodes;
  }

  ptrdiff_t idx = d_nodes[node].edges - d_nodes[0].edges;
  __syncthreads();
  d_graph->nodes[node].edges = &d_edges[idx];
  __syncthreads();
}

__global__ void calculate(dense_graph *d_graph,
                          connected_components *d_components,
                          component *d_comps, int *d_nodes, bool *d_used_nodes) {
  int node = blockIdx.x * blockDim.x * blockDim.y + threadIdx.x * blockDim.y +
             threadIdx.y;
  if (node >= d_graph->num_nodes)
    return;
  bool *used_nodes = d_used_nodes + node * d_graph->num_nodes;
  d_comps[node].num_nodes = 0;
  int *worklist = d_nodes + node * d_graph->num_nodes;
  int wl_len = 1, wl_pos = 0;
  worklist[0] = node;
  used_nodes[node] = true;
  while (wl_len) {
    int curr = worklist[wl_pos++];
    wl_len--;
    for (int i = 0; i < d_graph->nodes[curr].num_edges; i++) {
      int target = d_graph->nodes[curr].edges[i];
      if (target < node) { // not handled by this thread
        d_comps[node].num_nodes = 0;
        return;
      }
      if (used_nodes[target]) continue;
      used_nodes[target] = true;
      worklist[wl_pos + wl_len++] = target;
    }
  }
  d_comps[node].num_nodes = wl_pos;
  d_comps[node].nodes = d_nodes + node * d_graph->num_nodes;
  atomicInc(&d_components->num_components, d_graph->num_nodes);
}

extern "C" {
clock_t connected_components_thread_per_cc(dense_graph *graph,
                                           connected_components **out) {
  dense_graph *d_graph;
  dense_node *d_gnodes;
  dense_edge *d_edges;
  CHECK(hipMalloc((void **)&d_graph, sizeof(dense_graph)));
  CHECK(hipMalloc((void **)&d_gnodes, sizeof(dense_node) * graph->num_nodes));
  CHECK(hipMalloc((void **)&d_edges, MAX(sizeof(dense_edge) * graph->num_edges * 2, 1)));
  CHECK(hipMemcpy(d_graph, graph, sizeof(dense_graph), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_gnodes, graph->nodes, sizeof(dense_node) * graph->num_nodes, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_edges, graph->nodes->edges, sizeof(dense_edge) * graph->num_edges * 2, hipMemcpyHostToDevice));
  dim3 block, grid;
  block.x = 32;
  block.y = 32;
  grid.x = ceil((double)graph->num_nodes / block.x / block.y);
  grid.y = 1;
  populateGraph<<<grid, block>>>(d_graph, d_gnodes, d_edges, graph->nodes->edges);
  hipDeviceSynchronize();
  connected_components *d_components;
  CHECK(hipMalloc((void **)&d_components, sizeof(connected_components)));
  component *d_comps;
  CHECK(hipMalloc((void **)&d_comps, sizeof(component) * graph->num_nodes));
  int *d_nodes;
  CHECK(hipMalloc((void **)&d_nodes, sizeof(int) * graph->num_nodes * graph->num_nodes));
  bool *d_used_nodes;
  CHECK(hipMalloc((void **)&d_used_nodes, sizeof(bool) * graph->num_nodes * graph->num_nodes));
  clock_t start = clock();
  calculate<<<grid, block>>>(d_graph, d_components, d_comps, d_nodes, d_used_nodes);
  hipDeviceSynchronize();
  clock_t end = clock();
  // copy result back
  connected_components *components =
      (connected_components *)malloc(sizeof(connected_components));
  CHECK(hipMemcpy(components, d_components, sizeof(connected_components),
                   hipMemcpyDeviceToHost));

  components->components =
      (component *)malloc(sizeof(component) * graph->num_nodes);
  CHECK(hipMemcpy(components->components, d_comps,
                   sizeof(component) * graph->num_nodes, hipMemcpyDeviceToHost));

  int ind = 0;
  for (int i = 0; i < graph->num_nodes; i++) {
    if (!components->components[i].num_nodes)
      continue;
    node *d_nodes = components->components[i].nodes;
    components->components[ind].nodes =
        (int *)malloc(sizeof(int) * components->components[i].num_nodes);
    CHECK(hipMemcpy(components->components[ind].nodes, d_nodes,
                     sizeof(int) * components->components[i].num_nodes,
                     hipMemcpyDeviceToHost));
    components->components[ind].num_nodes = components->components[i].num_nodes;
    ind++;
  }
  // free gpu memory
  CHECK(hipFree(d_used_nodes));
  CHECK(hipFree(d_nodes));
  CHECK(hipFree(d_comps));
  CHECK(hipFree(d_components));
  CHECK(hipFree(d_edges));
  CHECK(hipFree(d_gnodes));
  CHECK(hipFree(d_graph));
  *out = components;
  return end - start;
}
}
