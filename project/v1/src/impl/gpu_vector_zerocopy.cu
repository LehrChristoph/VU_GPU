#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

extern "C" {
#include "impl.h"
}
#include "cuda_helpers.h"

#define MAX(a, b) ((a) > (b) ? (a) : (b))

/**
 * update references that still point to host memory
 */
__global__ void populateGraph_vec_zerocopy(dense_graph *d_graph, dense_node *d_nodes,
                                  dense_edge *d_edges, void *current_base,
                                  int *component_vector, int *base) {
  int node = blockIdx.x * blockDim.x * blockDim.y + threadIdx.x * blockDim.y +
             threadIdx.y;
  if (node >= d_graph->num_nodes)
    return;
  if (node == 0) {
    // update nodes ptr once
    d_graph->nodes = d_nodes;
    component_vector[d_graph->num_nodes] = 0;
  }

  component_vector[node] = -1;

  ptrdiff_t idx = d_nodes[node].edges - d_nodes[0].edges;
  __syncthreads();
  // update edge ptr of node using same index as before (ptr - baseptr)
  d_nodes[node].edges = &d_edges[idx];
  memset(&base[node * d_graph->num_nodes], 0, d_graph->num_nodes);
  __syncthreads();
}

__global__ void calculate_vec_zerocopy(dense_graph *d_graph, int *component_vector, int *base) {
  int node = blockIdx.x * blockDim.x * blockDim.y + threadIdx.x * blockDim.y +
             threadIdx.y;
  if (node >= d_graph->num_nodes)
    return;
  // processing stack
  base = &base[node * d_graph->num_nodes];
  int *stack = base;
  *(stack++) = node;

  while (base != stack) {
    int curr = *(--stack) & ~INT_MIN;
    // add node to cc
    int prev = atomicMax(&component_vector[curr], node);
    // already part of another, bigger cc
    if (prev > node) return;
    // skip nodes already in cc
    if (prev == node) continue;
    for (int i = 0; i < d_graph->nodes[curr].num_edges; i++) {
      int target = d_graph->nodes[curr].edges[i];
      if (target > node) {
        // not handled by this thread - thread with index of biggest node in
        // cc handles cc
        return;
      }
      // add node to worklist
      if (base[target] & INT_MIN) continue;
      *(stack++) = target | (*stack & INT_MIN);
      base[target] |= INT_MIN;
    }
  }
  // increment number of ccs
  atomicInc((unsigned int*) &component_vector[d_graph->num_nodes], d_graph->num_nodes);
}

extern "C" {
clock_t connected_components_vector_zerocopy(dense_graph *graph,
                                           connected_components **out) {
  dense_graph *h_graph;
  dense_node *h_gnodes;
  dense_edge *h_edges;
  CHECK(hipHostMalloc((void **)&h_graph, sizeof(dense_graph), hipHostMallocMapped));
  CHECK(hipHostMalloc((void **)&h_gnodes, sizeof(dense_node) * graph->num_nodes, hipHostMallocMapped));
  CHECK(hipHostMalloc((void **)&h_edges,
                   MAX(sizeof(dense_edge) * graph->num_edges * 2, 1), hipHostMallocMapped));
  memcpy(h_graph, graph, sizeof(dense_graph));
  memcpy(h_gnodes, graph->nodes, sizeof(dense_node) * graph->num_nodes);
  memcpy(h_edges, graph->nodes->edges, MAX(sizeof(dense_edge) * graph->num_edges * 2, 1));
  h_graph->nodes = h_gnodes;
  dense_graph *d_graph;
  dense_node *d_gnodes;
  dense_edge *d_edges;
  clock_t start = clock();
  CHECK(hipHostGetDevicePointer((void **)&d_graph, (void *)h_graph, 0));
  CHECK(hipHostGetDevicePointer((void **)&d_gnodes, (void *)h_gnodes, 0));
  CHECK(hipHostGetDevicePointer((void **)&d_edges, (void *)h_edges, 0));
  // allocate space for cc on device
  int *d_componentVector;
  int *result;
  // last is for num of components
  CHECK(hipHostMalloc((void **)&result, sizeof(int) * (graph->num_nodes + 1), hipHostMallocMapped));
  CHECK(hipHostGetDevicePointer((void **)&d_componentVector, (void *)result, 0));
  int *d_stack;
  CHECK(hipMalloc((void **)&d_stack,
                   sizeof(int) * graph->num_nodes * graph->num_nodes));
  dim3 block, grid;
  block.x = 32;
  block.y = 32;
  grid.x = ceil((double)graph->num_nodes / block.x / block.y);
  grid.y = 1;
  populateGraph_vec_zerocopy<<<grid, block>>>(d_graph, d_gnodes, d_edges,
                                     graph->nodes->edges, d_componentVector,
                                     d_stack);
  hipDeviceSynchronize();
  // doing calculation
  clock_t calcStart = clock();
  calculate_vec_zerocopy<<<grid, block>>>(d_graph, d_componentVector, d_stack);
  hipDeviceSynchronize();
  clock_t calcEnd = clock();
  // copy result back
  connected_components *components =
      (connected_components *)malloc(sizeof(connected_components));
  clock_t end = clock();
  components->num_components = result[graph->num_nodes];
  components->components =
      (component *)malloc(sizeof(component) * components->num_components);
  for (int i = 0; i < components->num_components; i++) {
    for (int j = 0; j < graph->num_nodes; j++) {
      bool free = true;
      for (int k = 0; k < i; k++) {
        if (components->components[k].num_nodes == result[j]) {
          free = false;
          break;
        }
      }
      if (free) {
        components->components[i].num_nodes = result[j];
        break;
      }
    }
  }
  int *nodes = (int *)malloc(sizeof(int) * graph->num_nodes);
  for (int i = 0; i < components->num_components; i++) {
      int num_nodes = 0;
      components->components[i].nodes = nodes;
      for (int j = 0; j < graph->num_nodes; j++) {
          if (result[j] == components->components[i].num_nodes) {
              num_nodes++;
              *(nodes++) = j;
          }
      }
      components->components[i].num_nodes = num_nodes;
  }
  components->single_node_list = true;

  // free gpu memory
  CHECK(hipFree(d_stack));
  CHECK(hipHostFree(result));
  *out = components;
  #ifdef BENCH_INCL_MEMCPY
  return end - start;
  #else
  return calcEnd - calcStart;
  #endif
}
}
